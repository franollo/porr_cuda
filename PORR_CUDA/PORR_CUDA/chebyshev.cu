#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include ""
#include "hip/hip_runtime.h"
#include "chebyshev.cuh"
#include "kernel.cu"
#include <string>
#include <stdlib.h>
#include <math.h>

#define Im1 0               // i - 1
#define I 1                 // i
#define Ip1 2               // i + 1


#define THREADS_PER_BLOCK 512


using namespace std;

double* chebyshev(int matrix_size, double** Ab, int s, int max_iter) {
	int N = matrix_size;
	dim3 threadsPerBlock(matrix_size);
	dim3 blocksPerGrid(1);
	if (matrix_size > 512) {
		threadsPerBlock.x = 512;
		blocksPerGrid.x = ceil(double(matrix_size) / double(threadsPerBlock.x));
	}

	double delta, x_2_norm, a, w_0, c, L, B, scalar_1, scalar_2;
	double *x_start, *A_vector, *b_vector, *w, *norm_vector;										//HOST	
	double *d_x, *d_scalar_1, *d_scalar_2, *d_temp_vector, *d_x_start, *d_A_vector, *d_b_vector;    //DEVICE
	int iteration, k;

	delta = 0.000001;       // accuracy
	x_2_norm = 0;           // second norm for stop criteria
	a = 100;                // alpha
	bool stop = false;      // stop criteria bool

	//ALLOCATE MEMORY ON HOST
	x_start = (double *)malloc(matrix_size * sizeof(double));
	A_vector = (double *)malloc(matrix_size * matrix_size * sizeof(double));
	b_vector = (double *)malloc(matrix_size * sizeof(double));
	w = (double *)malloc(matrix_size * 2 * sizeof(double));
	norm_vector = (double *)malloc(matrix_size * sizeof(double));

	//ALLOCATE MEMORY ON DEVICE
	hipMalloc((void **)&d_x, matrix_size * 3 * sizeof(double));
	hipMalloc((void **)&d_scalar_1, sizeof(double));
	hipMalloc((void **)&d_scalar_2, sizeof(double));
	hipMalloc((void **)&d_temp_vector, matrix_size * sizeof(double));
	hipMalloc((void **)&d_x_start, matrix_size * sizeof(double));
	hipMalloc((void **)&d_A_vector, matrix_size * matrix_size * sizeof(double));
	hipMalloc((void **)&d_b_vector, matrix_size * sizeof(double));


	B = Ab[0][0];

	//init x_start, find Beta, copy Ab to vectors
	for (int i = 0; i < matrix_size; i++) {
		x_start[i] = 0;
		b_vector[i] = Ab[i][matrix_size];
		if (Ab[i][i] > B) {
			B = Ab[i][i];
		}
		for (int j = 0; j < matrix_size; j++) {
			A_vector[i * N + j] = Ab[i][j];
		}
	}

	B = 2 * B;

	//COPY FROM HOST TO DEVICE
	hipMemcpy(d_x_start, x_start, matrix_size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_A_vector, A_vector, matrix_size * matrix_size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b_vector, b_vector, matrix_size * sizeof(double), hipMemcpyHostToDevice);

	//Step 0:
	iteration = 0;
	w_0 = (B - a) / (B + a);
	c = 2 / (B + a);
	L = 2 * (B + a) / (B - a);

	while (iteration < max_iter && stop == false) {
		//Step 1
		k = 0;

		copyVectorToMatRowKernel <<<blocksPerGrid, threadsPerBlock >>>(d_x_start, d_x, I, N);

		w[Im1] = 0;
		w[I] = w_0;

		while (iteration < max_iter) {
			//Step 2
			scalar_1 = c * (1 + w[I] * w[Im1]);
			scalar_2 = w[I] * w[Im1];
			hipMemcpy(d_scalar_1, &scalar_1, sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(d_scalar_2, &scalar_2, sizeof(double), hipMemcpyHostToDevice);

			//copies x(i) to temp_vector
			copyMatRowToVectorKernel <<<blocksPerGrid, threadsPerBlock>>>(d_x, d_temp_vector, I, N);

			//multiples A matrix in vector form with x(i) stored in temp_vector
			multMatrixVectorKernel <<<blocksPerGrid, threadsPerBlock>>>(d_A_vector, d_temp_vector, d_temp_vector, N);

			//calculates x(i+1) and sets x(i-1) and x(i)
			calculateXplus1Kernel <<<blocksPerGrid, threadsPerBlock>>>(d_x, d_temp_vector, d_b_vector, d_scalar_1, d_scalar_2, N);

			w[Im1] = w[I];
			w[I] = 1 / L - w[I];
			
			x_2_norm = 0;

			normKernel <<<blocksPerGrid, threadsPerBlock >> >(d_x, d_temp_vector, N);

			hipMemcpy(norm_vector, d_temp_vector, N * sizeof(double), hipMemcpyDeviceToHost);

			for (int i = 0; i < N; i++) {
				x_2_norm += norm_vector[i];
			}
			x_2_norm = sqrt(x_2_norm);
			if (x_2_norm < delta) {
				stop = true;
				break;
			}

			// Step 3
			iteration++;
			k++;
			if (k >= s) {
				copyMatRowToVectorKernel << <blocksPerGrid, threadsPerBlock >> >(d_x, d_x_start, I, N);
				break;
			}
		}
	}
	copyMatRowToVectorKernel << <blocksPerGrid, threadsPerBlock >> >(d_x, d_temp_vector, I, N);
	hipMemcpy(x_start, d_temp_vector, N * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_scalar_1);
	hipFree(d_scalar_2);
	hipFree(d_temp_vector);
	hipFree(d_x_start);
	hipFree(d_A_vector);
	hipFree(d_b_vector);

	free(A_vector);
	free(b_vector);
	free(w);
	free(norm_vector);
	return x_start;
}

void matrix_test() {

}



